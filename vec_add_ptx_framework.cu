#include <iostream>
#include <fstream>
#include <random>
#include <string>
#include <hip/hip_runtime.h>

// Function prototype
hipError_t launchKernel(const char* ptxCode, float* A, float* B, float* C, int N);

int main(int argc, char* argv[]) {
    int N = 100 * 1000 * 1000; // default value
    N = std::atoi(argv[1]);
    int size = N * sizeof(float);
    float *A, *B, *C;
    float *dA, *dB, *dC; // Device pointers

    // Allocate host memory
    A = new float[N];
    B = new float[N];
    C = new float[N];

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0, 100);

    for (int i = 0; i < N; i ++) {
        A[i] = dis(gen);
        B[i] = dis(gen);
    }

    // Allocate device memory
    hipMalloc(&dA, size);
    hipMalloc(&dB, size);
    hipMalloc(&dC, size);

    // Copy data from host to device
    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, size, hipMemcpyHostToDevice);

    // Load the PTX code from file
    std::ifstream ptxFileStream("vec_add_kernel.ptx");
    if (!ptxFileStream.is_open()) {
        std::cerr << "Failed to open PTX file." << std::endl;
        return 1;
    }
    std::string ptxCode((std::istreambuf_iterator<char>(ptxFileStream)), std::istreambuf_iterator<char>());
    if (ptxCode.empty()) {
        std::cerr << "PTX code is empty." << std::endl;
        return 1;
    }



    // Launch the kernel
    hipError_t cudaStatus = launchKernel(ptxCode.c_str(), dA, dB, dC, N);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    // Copy result back to host
    hipMemcpy(C, dC, size, hipMemcpyDeviceToHost);


    // Cleanup
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
