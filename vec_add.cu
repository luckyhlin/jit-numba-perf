
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <random>
#include <cstdlib>

__global__ void vec_add_kernel(float *a, float *b, float *result, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        result[index] = a[index] + b[index];
    }
}

int main(int argc, char* argv[]) {
    int N = 100 * 1000 * 1000; // default value
    N = std::atoi(argv[1]);
    int size = N * sizeof(float);

    float *hA = new float[N];
    float *hB = new float[N];
    float *dA, *dB, *dResult;
    hipMalloc((void **)&dA, size);
    hipMalloc((void **)&dB, size);
    hipMalloc((void **)&dResult, size);

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0, 100);

    for (int i = 0; i < N; i ++) {
        hA[i] = dis(gen);
        hB[i] = dis(gen);
    }

    // move data to device
    hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // cuda event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vec_add_kernel<<<blocksPerGrid, threadsPerBlock>>>(dA, dB, dResult, N);
    hipEventRecord(stop);

    // move result back to host
    hipMemcpy(hB, dB, size, hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Elapsed CUDA kernal time: " << milliseconds << "ms\n";

    // deallocate memory
    delete[] hA;
    delete[] hB;
    hipFree(dA);
    hipFree(dB);
    hipFree(dResult);

    return 0;
}
