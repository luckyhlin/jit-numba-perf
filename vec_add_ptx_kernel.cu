#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <iostream>
#include <string>

hipError_t launchKernel(const char* ptxCode, float* d_A, float* d_B, float* d_C, int N) {
    hipModule_t cuModule;
    hipFunction_t cuFunction;
    hipError_t res;

    // Initialize CUDA Driver API
    res = hipInit(0);
    if (res != hipSuccess) {
        std::cerr << "hipInit failed: res=" << res << std::endl;
        return hipErrorNotInitialized;
    }

    // Get a handle to the first CUDA device
    hipDevice_t cuDevice;
    res = hipDeviceGet(&cuDevice, 0);
    if (res != hipSuccess) {
        std::cerr << "hipDeviceGet failed: res=" << res << std::endl;
        return hipErrorNotInitialized;
    }

    // Create a context
    hipCtx_t cuContext;
    res = hipCtxCreate(&cuContext, 0, cuDevice);
    if (res != hipSuccess) {
        std::cerr << "hipCtxCreate failed: res=" << res << std::endl;
        return hipErrorNotInitialized;
    }

    // Load PTX from string
    res = hipModuleLoadData(&cuModule, ptxCode);
    if (res != hipSuccess) {
        std::cerr << "hipModuleLoadData failed: res=" << res << std::endl;
        hipCtxDestroy(cuContext);
        return hipErrorNotInitialized;
    }

    // Get a handle to the kernel function
    res = hipModuleGetFunction(&cuFunction, cuModule, "vectorAdd");
    if (res != hipSuccess) {
        std::cerr << "hipModuleGetFunction failed: res=" << res << std::endl;
        hipModuleUnload(cuModule);
        hipCtxDestroy(cuContext);
        return hipErrorNotInitialized;
    }

    // Set kernel parameters
    void* args[] = { &d_A, &d_B, &d_C, &N };

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    hipEventRecord(start);
    res = hipModuleLaunchKernel(cuFunction, blocksPerGrid, 1, 1, threadsPerBlock, 1, 1, 0, NULL, args, NULL);
    hipEventRecord(stop);
    if (res != hipSuccess) {
        std::cerr << "hipModuleLaunchKernel failed: res=" << res << std::endl;
        hipModuleUnload(cuModule);
        hipCtxDestroy(cuContext);
        return hipErrorLaunchFailure;
    }

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Elapsed CUDA kernal time: " << milliseconds << "ms\n";

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipModuleUnload(cuModule);
    hipCtxDestroy(cuContext);

    return hipSuccess;
}
